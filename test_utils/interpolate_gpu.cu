#include "hip/hip_runtime.h"  

#include "stdio.h"

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// input: features (n,c), idx (n,3), weight (n,3)
// output: probs (n,c)
__global__ void interpolate_kernel(int n, int c, int k_num, float *features,  float *weight,  int *idx, float *probs) {

    for(int i= blockIdx.x; i<n; i+=gridDim.x){
        for(int j=threadIdx.x; j<c;j+=blockDim.x){
            float tmp_prob = 0;
            for(int k=0; k<k_num;k++){
                int idd = idx[i*k_num + k];
                float wgt = weight[i*k_num + k];
                tmp_prob += features[idd*c+j]*wgt;      
            }
            probs[i*c+j] += tmp_prob;
            //printf("%f\t", tmp_prob);
        }
    }
}

extern "C" void interpolateLauncher(int n_host, int m_host, int c_host, float *features_host, int *idx_host, float *weight_host, float *probs_host){
    //int *n_dev, *c_dev;
    float *weight, *features, *probs;
    int *idx;
    hipError_t error;

    hipMalloc((void**)&weight, sizeof(float)* n_host*3);  
    hipMalloc((void**)&idx, sizeof(int)* n_host*3); 
    hipMalloc((void**)&features, sizeof(float)* m_host*c_host);  
    hipMalloc((void**)&probs, sizeof(float)* n_host*c_host);  

    hipMemcpy(weight, weight_host, sizeof(float)* n_host*3, hipMemcpyHostToDevice);
    hipMemcpy(idx, idx_host, sizeof(int)* n_host*3, hipMemcpyHostToDevice);
    hipMemcpy(features, features_host, sizeof(float)* m_host*c_host, hipMemcpyHostToDevice);
    hipMemcpy(probs, probs_host, sizeof(float)*n_host*c_host, hipMemcpyHostToDevice);

    dim3 grid(32768, 1, 1), block(c_host, 3, 1);
    interpolate_kernel<<<grid, block>>>(n_host, c_host, 3, features, weight, idx, probs);
    error = hipDeviceSynchronize();
    if(error != hipSuccess){
        printf("code: %d, reason: %s\n",error,hipGetErrorString(error));
    }


    hipMemcpy(probs_host, probs, sizeof(float)*n_host*c_host, hipMemcpyDeviceToHost);

    hipFree(weight);
    hipFree(features);
    hipFree(probs);
    hipFree(idx);
}


extern "C" void filterLauncher(int n_host, int c_host, int k_num, int iter_num, float *features_host, int *idx_host, float *weight_host, float *probs_host){
    //int *n_dev, *c_dev;
    float *weight, *features, *probs;
    int *idx;
    hipError_t error;

    hipMalloc((void**)&weight, sizeof(float)* n_host*k_num);  
    hipMalloc((void**)&idx, sizeof(int)* n_host*k_num); 
    hipMalloc((void**)&features, sizeof(float)* n_host*c_host);  
    hipMalloc((void**)&probs, sizeof(float)* n_host*c_host);  

    hipMemcpy(weight, weight_host, sizeof(float)* n_host*k_num, hipMemcpyHostToDevice);
    hipMemcpy(idx, idx_host, sizeof(int)* n_host*k_num, hipMemcpyHostToDevice);
    hipMemcpy(features, features_host, sizeof(float)* n_host*c_host, hipMemcpyHostToDevice);
    hipMemcpy(probs, probs_host, sizeof(float)*n_host*c_host, hipMemcpyHostToDevice);

    dim3 grid(32768, 1, 1), block(c_host, 1, 1);
    for(int i=0;i<iter_num;i++){
        interpolate_kernel<<<grid, block>>>(n_host, c_host, k_num, features, weight, idx, probs);
        error = hipDeviceSynchronize();
        if(error != hipSuccess){
            printf("code: %d, reason: %s\n",error,hipGetErrorString(error));
        }
    }

    hipMemcpy(probs_host, probs, sizeof(float)*n_host*c_host, hipMemcpyDeviceToHost);

    hipFree(weight);
    hipFree(features);
    hipFree(probs);
    hipFree(idx);
}
